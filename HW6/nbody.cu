#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------------- 
// CUDA code to compute minimun distance between n points
//
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <limits>
#include <float.h>

#define MAX_POINTS 1048576

// ---------------------------------------------------------------------------- 
// Kernel Function to compute distance between all pairs of points
// Input: 
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: D[0] = minimum distance
//

#define block_size 1024


// Working minimum_distance function
__device__ unsigned int finished_blocks = 0; 

__global__ void minimum_distance(float * X, float * Y, volatile float * D, int n) 
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int i, z;
    float dx, dy, temp_distance;    
    float minDist = FLT_MAX; 
    
    bool isLastBlockDone;
    
    __shared__ float block_local_minimums[block_size];

    if(idx < n - 1)
    {
        for(z = idx + 1; z<n; z++)
        {   
            dx = X[z] - X[idx];
            dy = Y[z] - Y[idx];

            temp_distance = sqrtf(dx * dx + dy * dy);   

            if(temp_distance < minDist)
            {
                minDist = temp_distance;
            }
        }

        block_local_minimums[threadIdx.x] = minDist;
    
        __syncthreads();

        // Compute the block local minimum
        int largest_index = (n % block_size);

        if(largest_index == 0)
        {
            largest_index = block_size;   
        }
        else
        {
            if(blockIdx.x != n/block_size)
            {
                largest_index = block_size;    
            }
        }

        for(i = 1; i<largest_index; i *= 2) 
        {
            if(threadIdx.x % (2 * i) == 0 && (threadIdx.x + i) < largest_index - 1)
            {
                if(block_local_minimums[threadIdx.x] > block_local_minimums[threadIdx.x + i])
                {
                    block_local_minimums[threadIdx.x] = block_local_minimums[threadIdx.x + i];  
                }

                __syncthreads();
            } 
        }

        if(threadIdx.x == 0)
        {
            D[blockIdx.x] = block_local_minimums[0];

            int value = atomicInc(&finished_blocks, gridDim.x);
            isLastBlockDone = (value == (gridDim.x - 1));
        }

        // Last thread in the list computes the global minimum and puts it in D[0]
        if(isLastBlockDone && threadIdx.x == 0)
        {
            int num_blocks = n / block_size + (n % block_size != 0);

            for(i = 1; i<num_blocks; i++)
            {
                if(D[0] > D[i])
                {
                    D[0] = D[i];                   
                }
            }
        }            
    }    
}


// ---------------------------------------------------------------------------- 
// Host function to compute minimum distance between points
// Input:
//	X: X[i] = x-coordinate of the ith point
//	Y: Y[i] = y-coordinate of the ith point
//	n: number of points
// Output: 
//	D: minimum distance
//
float minimum_distance_host(float * X, float * Y, int n) {
    float dx, dy, Dij, min_distance, min_distance_i;
    int i, j;
    dx = X[1]-X[0];
    dy = Y[1]-Y[0];
    min_distance = sqrtf(dx*dx+dy*dy);
    for (i = 0; i < n-1; i++) {
	for (j = i+1; j < i+2; j++) {
	    dx = X[j]-X[i];
	    dy = Y[j]-Y[i];
	    min_distance_i = sqrtf(dx*dx+dy*dy);
	}
	for (j = i+1; j < n; j++) {
	    dx = X[j]-X[i];
	    dy = Y[j]-Y[i];
	    Dij = sqrtf(dx*dx+dy*dy);
	    if (min_distance_i > Dij) min_distance_i = Dij;
	}
	if (min_distance > min_distance_i) min_distance = min_distance_i;
    }
    return min_distance;
}
// ---------------------------------------------------------------------------- 
// Print device properties
void print_device_properties() {
    int i, deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    printf("------------------------------------------------------------\n");
    printf("Number of GPU devices found = %d\n", deviceCount);
    for ( i = 0; i < deviceCount; ++i ) {
	hipGetDeviceProperties(&deviceProp, i);
	printf("[Device: %1d] Compute Capability %d.%d.\n", i, deviceProp.major, deviceProp.minor);
	printf(" ... multiprocessor count  = %d\n", deviceProp.multiProcessorCount); 
	printf(" ... max threads per multiprocessor = %d\n", deviceProp.maxThreadsPerMultiProcessor); 
	printf(" ... max threads per block = %d\n", deviceProp.maxThreadsPerBlock); 
	printf(" ... max block dimension   = %d, %d, %d (along x, y, z)\n",
		deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]); 
	printf(" ... max grid size         = %d, %d, %d (along x, y, z)\n",
		deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]); 
	printf(" ... warp size             = %d\n", deviceProp.warpSize); 
	printf(" ... clock rate            = %d MHz\n", deviceProp.clockRate/1000); 
    }
    printf("------------------------------------------------------------\n");
}
// ---------------------------------------------------------------------------- 
// Main program - initializes points and computes minimum distance 
// between the points
//
int main(int argc, char* argv[]) {

    // Host Data
    float * hVx;		// host x-coordinate array
    float * hVy;		// host y-coordinate array
    float * hmin_dist;		// minimum value on host

    // Device Data
    float * dVx;		// device x-coordinate array
    float * dVy;		// device x-coordinate array
    float * dmin_dist;		// minimum value on device

    // Device parameters
    int MAX_BLOCK_SIZE;		// Maximum number of threads allowed on the device
    int blocks;			// Number of blocks in grid
    int threads_per_block;	// Number of threads per block

    // Timing variables
    hipEvent_t start, stop;		// GPU timing variables
    struct timespec cpu_start, cpu_stop; // CPU timing variables
    float time_array[10]; 

    // Other variables
    int i, size, num_points; 
    float min_distance, sqrtn;
    int seed = 0;

    // Print device properties
    print_device_properties(); 

    // Get device information and set device to use
    int deviceCount;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
	hipSetDevice(0); 
	hipGetDeviceProperties(&deviceProp, 0);
	MAX_BLOCK_SIZE = deviceProp.maxThreadsPerBlock;
    } else {
	printf("Warning: No GPU device found ... results may be incorrect\n");
    }

    // Timing initializations
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Check input
    if (argc != 2) {
	printf("Use: %s <number of points>\n", argv[0]);  
	exit(0);
    }
    if ((num_points = atoi(argv[argc-1])) < 2) {
	printf("Minimum number of points allowed: 2\n");
	exit(0);
    } 
    if ((num_points = atoi(argv[argc-1])) > MAX_POINTS) {
	printf("Maximum number of points allowed: %d\n", MAX_POINTS);
	exit(0);
    } 

    // Allocate host coordinate arrays 
    size = num_points * sizeof(float); 
    hVx = (float *) malloc(size); 
    hVy = (float *) malloc(size);
    hmin_dist = (float *) malloc(size);

    // Initialize points
    srand48(seed);                                // UNCOMMENT THIS UNCOMMENT THIS UNCOMMENT THIS!!!
    sqrtn = (float) sqrt(num_points); 
    for (i = 0; i < num_points; i++) {
	hVx[i] = sqrtn * (float)drand48();
	hVy[i] = sqrtn * (float)drand48();
    }

    // Allocate device coordinate arrays
    hipMalloc(&dVx, size);
    hipMalloc(&dVy, size);
    hipMalloc(&dmin_dist, size);

    // Copy coordinate arrays from host memory to device memory 
    hipEventRecord( start, 0 ); 

    hipMemcpy(dVx, hVx, size, hipMemcpyHostToDevice);
    hipMemcpy(dVy, hVy, size, hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[0]), start, stop);

    // Invoke kernel
    hipEventRecord( start, 0 ); 

    // Custom variables to create blocks
    int num_blocks = num_points / (block_size) + ((num_points % (block_size)) != 0);

    //
    // Kernel function invocation
    //
    minimum_distance<<<num_blocks, block_size>>>(dVx, dVy, dmin_dist, num_points);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[1]), start, stop);

    // Copy result from device memory to host memory 
    hipEventRecord( start, 0 ); 

    hipMemcpy(hmin_dist, dmin_dist, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&(time_array[2]), start, stop);

    // Compute minimum distance on host to check device computation
    clock_gettime(CLOCK_REALTIME, &cpu_start);

    min_distance = minimum_distance_host(hVx, hVy, num_points); 

    clock_gettime(CLOCK_REALTIME, &cpu_stop);
    time_array[3] = 1000*((cpu_stop.tv_sec-cpu_start.tv_sec)                    
	    +0.000000001*(cpu_stop.tv_nsec-cpu_start.tv_nsec));

    // Print results
    printf("Number of Points    = %d\n", num_points); 
    printf("GPU Host-to-device  = %f ms \n", time_array[0]);
    printf("GPU Device-to-host  = %f ms \n", time_array[2]);
    printf("GPU execution time  = %f ms \n", time_array[1]);
    printf("CPU execution time  = %f ms\n", time_array[3]);
    printf("Min. distance (GPU) = %e\n", hmin_dist[0]);
    printf("Min. distance (CPU) = %e\n", min_distance);
    printf("Relative error      = %e\n", fabs(min_distance-hmin_dist[0])/min_distance);


    // Free device memory 
    hipFree(dVx);
    hipFree(dVy);
    hipFree(dmin_dist);

    // Free host memory 
    free(hVx);
    free(hVy);
}  
